#include "hip/hip_runtime.h"
#include "imglib/format_type.h"

namespace imglib
{

template<typename _OutPixelType, typename _OutFormat, typename _InPixelType, typename _InFormat>
__device__ _OutPixelType translatePixel(const _InPixelType &input)
{
    return _OutPixelType();
}

/////////////////////////////
//float3 output
/////////////////////////////
template<>
__device__ float3 translatePixel<float3, Format_RGB, uchar3, Format_RGB>(const uchar3 &pixel)
{
    return make_float3(pixel.x, pixel.y, pixel.z);
}

template<>
__device__ float3 translatePixel<float3, Format_RGB, uchar3, Format_BGR>(const uchar3 &pixel)
{
    return make_float3(pixel.z, pixel.y, pixel.x);
}

template<>
__device__ float3 translatePixel<float3, Format_RGB, uchar4, Format_RGBA>(const uchar4 &pixel)
{
    return make_float3(pixel.x, pixel.y, pixel.z);
}

template<>
__device__ float3 translatePixel<float3, Format_RGB, uchar4, Format_BGRA>(const uchar4 &pixel)
{
    return make_float3(pixel.z, pixel.y, pixel.x);
}

template<>
__device__ float3 translatePixel<float3, Format_BGR, uchar3, Format_RGB>(const uchar3 &pixel)
{
    return make_float3(pixel.z, pixel.y, pixel.x);
}

template<>
__device__ float3 translatePixel<float3, Format_BGR, uchar3, Format_BGR>(const uchar3 &pixel)
{
    return make_float3(pixel.x, pixel.y, pixel.z);
}

template<>
__device__ float3 translatePixel<float3, Format_BGR, uchar4, Format_RGB>(const uchar4 &pixel)
{
    return make_float3(pixel.z, pixel.y, pixel.x);
}

template<>
__device__ float3 translatePixel<float3, Format_BGR, uchar4, Format_BGRA>(const uchar4 &pixel)
{
    return make_float3(pixel.x, pixel.y, pixel.z);
}

/////////////////////////////
//float4 output
/////////////////////////////
template<>
__device__ float4 translatePixel<float4, Format_RGBA, uchar3, Format_RGB>(const uchar3 &pixel)
{
    return make_float4(pixel.x, pixel.y, pixel.z, 1.0f);
}

template<>
__device__ float4 translatePixel<float4, Format_RGBA, uchar3, Format_BGR>(const uchar3 &pixel)
{
    return make_float4(pixel.z, pixel.y, pixel.x, 1.0f);
}

template<>
__device__ float4 translatePixel<float4, Format_RGBA, uchar4, Format_RGBA>(const uchar4 &pixel)
{
    return make_float4(pixel.x, pixel.y, pixel.z, pixel.w);
}

template<>
__device__ float4 translatePixel<float4, Format_RGBA, uchar4, Format_BGRA>(const uchar4 &pixel)
{
    return make_float4(pixel.z, pixel.y, pixel.x, pixel.w);
}

template<>
__device__ float4 translatePixel<float4, Format_BGRA, uchar3, Format_RGB>(const uchar3 &pixel)
{
    return make_float4(pixel.z, pixel.y, pixel.x, 1.0f);
}

template<>
__device__ float4 translatePixel<float4, Format_BGRA, uchar3, Format_BGR>(const uchar3 &pixel)
{
    return make_float4(pixel.x, pixel.y, pixel.z, 1.0f);
}

template<>
__device__ float4 translatePixel<float4, Format_BGRA, uchar4, Format_RGB>(const uchar4 &pixel)
{
    return make_float4(pixel.z, pixel.y, pixel.x, pixel.w);
}

template<>
__device__ float4 translatePixel<float4, Format_BGRA, uchar4, Format_BGRA>(const uchar4 &pixel)
{
    return make_float4(pixel.x, pixel.y, pixel.z, pixel.w);
}

}//namespace imglib