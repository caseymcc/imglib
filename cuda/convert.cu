#include "hip/hip_runtime.h"
#include "imglib/cuda/convert.h"
#include "imglib/format_type.h"
#include "imglib/cuda/util.h"

#include "translate.cu"

namespace imglib
{
namespace cuda
{

//template<typename _Format>
//__device__ __host__ int formatChannels(){return 0;}
//
//template<>
//__device__ __host__ int formatChannels<Format_Binary>(){return 1;}
//template<>
//__device__ __host__ int formatChannels<Format_GreyScale>(){return 1;}
//template<>
//__device__ __host__ int formatChannels<Format_RA>(){return 2;}
//template<>
//__device__ __host__ int formatChannels<Format_RGB>(){return 3;}
//template<>
//__device__ __host__ int formatChannels<Format_RGBA>(){return 4;}
//template<>
//__device__ __host__ int formatChannels<Format_RGBP>(){return 3;}
//template<>
//__device__ __host__ int formatChannels<Format_RGBAP>(){return 4;}
//template<>
//__device__ __host__ int formatChannels<Format_BGR>(){return 3;}
//template<>
//__device__ __host__ int formatChannels<Format_BGRA>(){return 4;}
//template<>
//__device__ __host__ int formatChannels<Format_BGRP>(){return 3;}
//template<>
//__device__ __host__ int formatChannels<Format_BGRAP>(){return 4;}

//template<typename _OutputFormat, typename _InputType, typename _InputFormat>
//__global__ void convertNormMean_NearestNeighbor_Plane(_InputType *input, int inputWidth, float *output, int outputWidth, int outputHeight, float2 scale, float multiplier, float minValue, const float3 mean, const float3 stdDev)
__global__ void convertNormMean_NearestNeighbor_Plane(uchar3 *input, int inputWidth, float *output, int outputWidth, int outputHeight, float2 scale, float multiplier, float minValue, const float3 mean, const float3 stdDev)
{
    const int x=(blockIdx.x*blockDim.x)+threadIdx.x;
    const int y=(blockIdx.y*blockDim.y)+threadIdx.y;

    if((x >= outputWidth) || (y >= outputHeight))
        return;

//    const int outputChannels=_OutputFormat::channels;
    const int outputChannels=3;

    const int outputPlaneStride=outputWidth*outputHeight;
    const int outputPos=(y*outputWidth)+x;

    const int dx=((float)x*scale.x);
    const int dy=((float)y*scale.y);

    //const _InputType pixel=input[dy*inputWidth+dx];
    const uchar3 pixel=input[dy*inputWidth+dx];

//    const float3 outPixel=translatePixel<float3, _OutputFormat, _InputType, _InputFormat>(pixel);
    const float3 outPixel=make_float3(pixel.x, pixel.y, pixel.z);
    
    output[(outputPlaneStride*0)+outputPos]=((outPixel.x*multiplier+minValue)-mean.x)/stdDev.x;
    output[(outputPlaneStride*1)+outputPos]=((outPixel.y*multiplier+minValue)-mean.y)/stdDev.y;
    output[(outputPlaneStride*2)+outputPos]=((outPixel.z*multiplier+minValue)-mean.z)/stdDev.z;
}

//template<typename _OutputFormat, typename _InputType, typename _InputFormat>
//__global__ void convertNormMean_NearestNeighbor_Packed(_InputType *input, int inputWidth, float *output, int outputWidth, int outputHeight, float2 scale, float multiplier, float minValue, const float3 mean, const float3 stdDev)
__global__ void convertNormMean_NearestNeighbor_Packed(uchar3 *input, int inputWidth, float *output, int outputWidth, int outputHeight, float2 scale, float multiplier, float minValue, const float3 mean, const float3 stdDev)
{
    const int x=(blockIdx.x*blockDim.x)+threadIdx.x;
    const int y=(blockIdx.y*blockDim.y)+threadIdx.y;

    if((x >= outputWidth) || (y >= outputHeight))
        return;

//    const int outputChannels=_OutputFormat::channels;
    const int outputChannels=3;

    const int outputStride=outputWidth*outputChannels;
    const int outputPos=(y*outputStride)+(x*outputChannels);

    const int dx=((float)x*scale.x);
    const int dy=((float)y*scale.y);

//    const _InputType pixel=input[dy*inputWidth+dx];
    const uchar3 pixel=input[dy*inputWidth+dx];

//    const float3 outPixel=translatePixel<float3, _OutputFormat, _InputType, _InputFormat>(pixel);
    const float3 outPixel=make_float3(pixel.x, pixel.y, pixel.z);
    
    output[outputPos]=((outPixel.x*multiplier+minValue)-mean.x)/stdDev.x;
    output[outputPos+1]=((outPixel.y*multiplier+minValue)-mean.y)/stdDev.y;
    output[outputPos+2]=((outPixel.z*multiplier+minValue)-mean.z)/stdDev.z;
}

//template<typename _InputType, typename _InputFormat>
//hipError_t convertNormMean_NearestNeighbor_Input(const dim3 &blockDim, const dim3 &gridDim, hipStream_t stream,
//    _InputType *input, size_t inputWidth, 
//    float *output, Format outputFormat, size_t outputWidth, size_t outputHeight, const float2 &scale, float multiplier, float minValue, const float3 &mean, const float3 &stdDev)
hipError_t convertNormMean_NearestNeighbor_Input(const dim3 &blockDim, const dim3 &gridDim, hipStream_t stream,
    uchar3 *input, size_t inputWidth, 
    float *output, Format outputFormat, size_t outputWidth, size_t outputHeight, const float2 &scale, float multiplier, float minValue, const float3 &mean, const float3 &stdDev)
{
    if(outputFormat == Format::RGB)
//        convertNormMean_NearestNeighbor_Packed<Format_RGB, _InputType, _InputFormat><<<gridDim, blockDim, 0, stream>>>(input, (int)inputWidth, output, (int)outputWidth, (int)outputHeight, scale, multiplier, minValue, mean, stdDev);
//        convertNormMean_NearestNeighbor_Packed<<<gridDim, blockDim, 0, stream>>>(input, (int)inputWidth, output, (int)outputWidth, (int)outputHeight, scale, multiplier, minValue, mean, stdDev);
        convertNormMean_NearestNeighbor_Packed<<<gridDim, blockDim>>>(input, (int)inputWidth, output, (int)outputWidth, (int)outputHeight, scale, multiplier, minValue, mean, stdDev);
    else if(outputFormat == Format::RGBP)
//        convertNormMean_NearestNeighbor_Plane<Format_RGBP, _InputType, _InputFormat><<<gridDim, blockDim, 0, stream>>>(input, (int)inputWidth, output, (int)outputWidth, (int)outputHeight, scale, multiplier, minValue, mean, stdDev);
        convertNormMean_NearestNeighbor_Plane<<<gridDim, blockDim, 0, stream>>>(input, (int)inputWidth, output, (int)outputWidth, (int)outputHeight, scale, multiplier, minValue, mean, stdDev);
//    else if(outputFormat == Format::BGR)
//        convertNormMean_NearestNeighbor_Packed<Format_BGR, _InputType, _InputFormat><<<gridDim, blockDim, 0, stream>>>(input, (int)inputWidth, output, (int)outputWidth, (int)outputHeight, scale, multiplier, minValue, mean, stdDev);
//    else if(outputFormat == Format::BGRP)
//        convertNormMean_NearestNeighbor_Plane<Format_BGRP, _InputType, _InputFormat><<<gridDim, blockDim, 0, stream>>>(input, (int)inputWidth, output, (int)outputWidth, (int)outputHeight, scale, multiplier, minValue, mean, stdDev);
    else
    {   
        assert(false);//not implemented
        return hipErrorInvalidValue;
    }

    return hipGetLastError();
}

hipError_t convertNormMean_NearestNeighbor(void *input, Format inputFormat, size_t inputWidth, size_t inputHeight,
    float *output, Format outputFormat, size_t outputWidth, size_t outputHeight, 
    const float2 &range, const float3& mean, const float3& stdDev, hipStream_t stream)
{
    if(!input || !output)
        return hipErrorInvalidDevicePointer;

    if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
        return hipErrorInvalidValue;

    const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
                                float(inputHeight) / float(outputHeight) );

    const float multiplier = (range.y - range.x) / 255.0f;
    
    // launch kernel
    const dim3 blockDim(8, 8);
    const dim3 gridDim(divCeil(outputWidth,blockDim.x), divCeil(outputHeight,blockDim.y));

    if(inputFormat == Format::RGB)
        return convertNormMean_NearestNeighbor_Input(blockDim, gridDim, stream, (uchar3 *)input, inputWidth, output, outputFormat, outputWidth, outputHeight, scale, multiplier, range.x, mean, stdDev);
//        return convertNormMean_NearestNeighbor_Input<uchar3, Format_RGB>(blockDim, gridDim, stream, (uchar3 *)input, inputWidth, output, outputFormat, outputWidth, outputHeight, scale, multiplier, range.x, mean, stdDev);
//    else if(inputFormat == Format::RGBA)
//        return convertNormMean_NearestNeighbor_Input<uchar4, Format_RGBA>(blockDim, gridDim, stream, (uchar4 *)input, inputWidth, output, outputFormat, outputWidth, outputHeight, scale, multiplier, range.x, mean, stdDev);
//    else if(inputFormat == Format::BGR)
//        return convertNormMean_NearestNeighbor_Input<uchar3, Format_BGR>(blockDim, gridDim, stream, (uchar3 *)input, inputWidth, output, outputFormat, outputWidth, outputHeight, scale, multiplier, range.x, mean, stdDev);
//    else if(inputFormat == Format::BGRA)
//        return convertNormMean_NearestNeighbor_Input<uchar4, Format_BGRA>(blockDim, gridDim, stream, (uchar4 *)input, inputWidth, output, outputFormat, outputWidth, outputHeight, scale, multiplier, range.x, mean, stdDev);

    assert(false);//not implemented
    return hipErrorInvalidValue;
}

}//namespace cuda
}//namespace imglib